#include "hip/hip_runtime.h"
/* 
 * pthreaded hw5, written by Adam Tygart abd Ryan Hershberger
 * Could be further optimized by pipelining read operations and not cyclically creating/destroying child threads
 */
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
/*
 * Length of "lines changes with every protein"
 * Thanks to wikipedia for the following pseudocode:
 * function LCSLength(X[1..m], Y[1..n])
 *     C = array(0..m, 0..n)
 *     for i := 0..m
 *        C[i,0] = 0
 *     for j := 0..n
 *        C[0,j] = 0
 *     for i := 1..m
 *         for j := 1..n
 *             if X[i] = Y[j]
 *                 C[i,j] := C[i-1,j-1] + 1
 *             else:
 *                 C[i,j] := max(C[i,j-1], C[i-1,j])
 *     return C[m,n]
 */

FILE *f;
int comp_count;
int offset = 0;

#ifndef NUM_THREADS
#define NUM_THREADS 1000
#endif

#ifndef WORK_UNIT
#define WORK_UNIT 100
#endif

#define QUEUE_SIZE NUM_THREADS*WORK_UNIT

/*
 * Calculate the LCS of the two strings.
 */
__device__ int MCSLength(char *str1, int len1, char* str2, int len2) {
	int** arr = (int**) malloc(sizeof(int*)*(len1+1));
	if ( arr == 0 ) {
		printf("Couldn't allocate memory for the MCS array\n");
	}
	int i, j, local_max = 0;
	for (i = 0; i <= len1; i++) {
		arr[i] = (int*)malloc((len2+1) *sizeof(int));
		if ( arr[i] == 0 ) {
			printf("Couldn't allocate memory for the MCS subarray\n");
		}
	}
	for (i = 1; i <= len1; i++) {
		for (j = 1; j <= len2; j++) {
			if (str1[i-1] == str2[j-1]) {
				arr[i][j] = arr[i-1][j-1] + 1;
				if (arr[i][j] > local_max)
					local_max = arr[i][j];
			}

		}
	}
	for (i = 0; i <= len1; i++)
		free(arr[i]);
	free(arr);
	return local_max;
}

/*
 * Read file, char by char. headers start with '>' or ';', ignore until newline.
 * read "gene" until we reach the next header. return int of num of chars in buff[i]
 */
int readLine(char **buff, int i) {
	int readchars = 0;
	int commentline = 0, startedgene = 0;
	int buffStepSize = 4000;
	int buffSize = 4000;
	buff[i] = (char*)malloc(sizeof(char)*buffSize);
	char c;
	do {
		if (((readchars) >= buffSize) && (buffSize != 0)) {
			buffSize += buffStepSize;
			char* temp_buff = (char*)realloc(buff[i],sizeof(char)*buffSize);
			buff[i] = temp_buff;
		}
		if (buff[i] == 0) {
			printf("Couldn't allocate memory for the buffer\n");
			exit(-2);
		}
		c = fgetc(f);
		switch (c) {
			case '\n':
				commentline = 0;
				break;
			case ';':
			case '>':
				commentline = 1;
				if (startedgene == 1) {
					long curr = ftell(f);
					fseek(f, curr-1, SEEK_SET);
					return readchars;
				}
				break;
			default:
				if ( commentline == 0 ) {
					startedgene = 1;
					if (c != EOF)
						buff[i][readchars++] = c;
				}
		}
	} while (c != EOF);
	return readchars;
}

/*
 * Is the worker function for a thread, calculate your chunk of the global data, calculate the MCS of each pair, copy the counts off to the global counts once locked
 */
__global__ void threaded_count( int* dev_counts, char** dev_queue, int* dev_lens, int perThread, int totalThreads) {
	int local_counts[QUEUE_SIZE/NUM_THREADS/2];
	int local_count = 0;
	int startPos = ((int) 0) * (QUEUE_SIZE/NUM_THREADS);
	int endPos = startPos + (QUEUE_SIZE/NUM_THREADS);

	int i, j;
	for (i = 0; i < QUEUE_SIZE/NUM_THREADS/2; i++) {
		local_counts[i] = 0;
		j = startPos + (i*2);
		if ((dev_lens[j] != 0) && (dev_lens[j+1] != 0)) {
			local_counts[i] = MCSLength(dev_queue[j], dev_lens[j], dev_queue[j+1], dev_lens[j+1]);
			local_count++;
		}
		else
			break;
	}
	for (i = 0; i < QUEUE_SIZE/NUM_THREADS/2; i++) {
		dev_counts[(offset/2) + (startPos/2) + i] = local_counts[i];
	}
	comp_count += local_count;
}

/*
 * Take a file-name on the command line, open it and read portions of the file at a time. start threads to calcluate MCS. Find the max and average MCSs
 */
int main(int argc, char* argv[]) {
	if (argc != 2 ) {
		printf("Please specify a file on the command line\n");
		exit(-1);
	}
	f = fopen(argv[1],"r");
	if ( f == 0 ) {
		printf("Couldn't open file\n");
		exit(-1);
	}
	char **queue;
	int *lens;
	int *counts;
	char **dev_queue;
	int *dev_lens;
	int *dev_counts;
	//pthread
	int i, rc;
	void *status;
	int perThread = WORK_UNIT;
	int totalSize = QUEUE_SIZE;
	int size = NUM_THREADS;
	int numThreadsPerBlock = 100;
	int numBlocks = size / numThreadsPerBlock;
	int totalThreads = numThreadsPerBlock * numBlocks;
	
	do {
		queue = (char**)malloc(sizeof(char*)*QUEUE_SIZE);
		hipMalloc((void**)&dev_queue, sizeof(char*)*QUEUE_SIZE);

		lens = (int*)calloc(sizeof(int),QUEUE_SIZE);
		hipMalloc((void**)&dev_lens, sizeof(int)*QUEUE_SIZE);

		int *temp_counts = (int*) realloc(counts, (QUEUE_SIZE + offset)/2 * sizeof(int));
		if (( queue == 0 ) || (lens == 0) || (temp_counts == 0)) {
			printf("Couldn't allocate memory for the work queues\n");
			exit(-1);
		}
		counts = temp_counts;
		for (i = 0; i < QUEUE_SIZE; i++) {
			lens[i] = readLine(queue, i);
			if (( queue[i] == 0 )) {
				printf("Couldn't allocate memory for the work subqueues\n");
				exit(-1);
			}
			hipMalloc((void*)&(dev_queue[i]), (lens[i])*sizeof(char));
			hipMemcpy(dev_queue[i], queue[i], lens[i]*sizeof(char), hipMemcpyHostToDevice);
		}
		hipMemcpy(dev_lens, lens, QUEUE_SIZE*sizeof(int), hipMemcpyHostToDevice);

		hipMalloc((void**)&dev_counts, (QUEUE_SIZE*sizeof(int))/2);
		hipMemset( dev_counts, 0, (QUEUE_SIZE*sizeof(int))/2);

		dim3 dimGrid(numBlocks);
		dim3 dimBlock(numThreadsPerBlock);
		threaded_count<<< dimGrid, dimBlock >>>(dev_counts, dev_queue, dev_lens, perThread, totalThreads);
		hipDeviceSynchronize();
		int* temp = (int*) malloc(sizeof(int)*QUEUE_SIZE/2);
		hipMemcpy(temp, dev_counts, (QUEUE_SIZE*sizeof(int))/2, hipMemcpyDeviceToHost);
		for (i = 0; i < QUEUE_SIZE/2; i++)
			counts[offset+i] = temp[i];

		for (i = 0; i < QUEUE_SIZE; i++) {
			hipFree(dev_queue[i]);
			free(queue[i]);
		}
		hipFree(dev_counts);
		free(temp);
		hipFree(dev_queue);
		free(queue);
		hipFree(dev_lens);
		free(lens);
		offset += QUEUE_SIZE;
	} while (!feof(f));
	unsigned long total = 0;
	int longest = 0, longest_loc = -1;
	for (i = 0; i < comp_count; i++) {
		total += counts[i];
		if (counts[i] > longest) {
			longest = counts[i];
			longest_loc = i;
		}
	}

	printf("Longest LCS: %d, is the %dth pair in the file\n", longest, longest_loc);
	printf("Average: %Lf\n",((long double) total)/comp_count);
	fclose(f);
	free(counts);
	pthread_exit(NULL);
	return 0;
}

